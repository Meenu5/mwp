#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <hip/hip_runtime.h>

void initialize(int *menacc, int *womenacc, int *menpre, int *womenlock, int n) {
    int i;
    for(i=0; i<=n; i++) {
        menacc[i] = -1;
        womenacc[i] = -1;
        menpre[i] = 1;
        womenlock[i] = 0;
    }
}

__global__ void stable_matching(int n, int *d_men, int *d_women,
        int *d_menacc, int *d_womenacc, int *d_menpre, int *d_matched, int *d_matched_, int *d_womenlock) {
    int j = threadIdx.x + 1, idx, ct=0;
    while(1) {
        __syncthreads();
        if(*d_matched_ == 0) break;
        if(*d_matched_ == 1 && j <= n && d_menacc[j] == -1) {
            idx = d_men[j*(n+1) + d_menpre[j]];
            *d_matched = 0;
            // locking mechanism
            bool isSet = false;
            do {
                if(isSet = atomicCAS(&d_womenlock[idx], 0, 1) == 0) {
                    if(d_womenacc[idx] == -1) {
                        d_womenacc[idx] = j;
                        d_menacc[j] = idx;
                    }
                    else if(d_women[idx*(n+1) + d_womenacc[idx]] > d_women[idx*(n+1) + j]) {
                        d_menacc[d_womenacc[idx]] = -1;
                        d_menacc[j] = idx;
                        d_womenacc[idx] = j;
                    }
                }
                if(isSet) {
                    atomicCAS(&d_womenlock[idx], 1, 0);
                }
            } while(!isSet);
            d_menpre[j]++;
        }
        __syncthreads();
        if(j == 1 && *d_matched == 1) {
            *d_matched_ = 0;
        }
        else if(j == 1 && *d_matched == 0) {
            *d_matched = 1;
        }
        ct++;
    }
    __syncthreads();
}

int main()
{
    int n,i,j,k;
    int *d_matched, *d_matched_;
    int *men, *women;
    int *menacc, *womenacc, *menpre, *womenlock;
    int *d_men, *d_women;
    int *d_menacc, *d_womenacc, *d_menpre, *d_womenlock;
    clock_t beg, end;
    double time_taken;

    scanf("%d",&n);
    men = (int *) malloc((n+1)*(n+1)*sizeof(int));
    women = (int *) malloc((n+1)*(n+1)*sizeof(int));
    menacc = (int *) malloc((n+1)*sizeof(int));
    womenacc = (int *) malloc((n+1)*sizeof(int));
    womenlock = (int *) malloc((n+1)*sizeof(int));
    menpre = (int *) malloc((n+1)*sizeof(int));

    hipMalloc(&d_men, (n+1)*(n+1)*sizeof(int));
    hipMalloc(&d_women, (n+1)*(n+1)*sizeof(int));
    hipMalloc(&d_menacc, (n+1)*sizeof(int));
    hipMalloc(&d_womenacc, (n+1)*sizeof(int));
    hipMalloc(&d_womenlock, (n+1)*sizeof(int));
    hipMalloc(&d_menpre, (n+1)*sizeof(int));
    hipMalloc(&d_matched, sizeof(int));
    hipMalloc(&d_matched_, sizeof(int));

    initialize(menacc, womenacc, menpre, womenlock, n);

    beg = clock();
    for(i=1; i<=n; i++) {
        for(j=0; j<=n; j++) {
            scanf("%d", &men[i*(n+1) + j]);
        }
    }

    for(i=1; i<=n; i++) {
        for(j=0; j<=n; j++) {
            scanf("%d", &k);
            women[i*(n+1) + k] = j;
        }
    }
    end = clock();
    time_taken = ((double)(end-beg) * 1000000)/CLOCKS_PER_SEC;
    printf("read time : %f us, ", time_taken);

    hipMemcpy(d_men, men, (n+1)*(n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_women, women, (n+1)*(n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_menacc, menacc, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_womenlock, womenlock, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_womenacc, womenacc, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_menpre, menpre, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    int matched = 1;
    hipMemcpy(d_matched, &matched, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matched_, &matched, sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    stable_matching <<< 1, n >>>(n, d_men, d_women, d_menacc, d_womenacc, d_menpre, d_matched, d_matched_, d_womenlock);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(menacc, d_menacc, (n+1)*sizeof(int), hipMemcpyDeviceToHost);
    printf("compute time : %f us\n", milliseconds*1000);

    for(j=1;j<=n;j++)
        printf("%d %d\n", j, menacc[j]);

    free(men); free(women);
    free(menacc); free(womenacc); free(menpre); free(womenlock);
    hipFree(&d_men); hipFree(&d_women); hipFree(&d_matched); hipFree(&d_matched_);
    hipFree(&d_menacc); hipFree(&d_womenacc); hipFree(&d_menpre); hipFree(&d_womenlock);

    return 0;
}
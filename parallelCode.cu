#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <hip/hip_runtime.h>

void initialize(int *menacc, int *womenacc, int *menpre, int *womenlock, int n) {
    int i;
    for(i=0; i<=n; i++) {
        menacc[i] = -1;
        womenacc[i] = -1;
        menpre[i] = 1;
        womenlock[i] = 0;
    }
}

__global__ void stable_matching(int n, int *d_men, int *d_women,
        int *d_menacc, int *d_womenacc, int *d_menpre, int *d_matched, int *d_womenlock) {
    int j = threadIdx.x + 1, idx;
    idx = d_men[j*(n+1) + d_menpre[j]];
    if(j <= n && d_menacc[j] == -1) {
        *d_matched = 0;
        // locking mechanism
        bool isSet = false;
        do {
            if(isSet = atomicCAS(&d_womenlock[idx], 0, 1) == 0) {
                if(d_womenacc[idx] == -1) {
                    d_womenacc[idx] = j;
                    d_menacc[j] = idx;
                }
                else if(d_women[idx*(n+1) + d_womenacc[idx]] > d_women[idx*(n+1) + j]) {
                    d_menacc[d_womenacc[idx]] = -1;
                    d_menacc[j] = idx;
                    d_womenacc[idx] = j;
                }
            }
            if(isSet) {
                atomicCAS(&d_womenlock[idx], 1, 0);
            }
        } while(!isSet);
        d_menpre[j]++;
    }
}

// driver function to utilize CUDA dynamic parallelism
__global__ void driver_function(int n, int *d_men, int *d_women,
        int *d_menacc, int *d_womenacc, int *d_menpre, int *d_matched, int *d_womenlock) {
    *d_matched = 0;
    while(!(*d_matched)) {
        *d_matched = 1;
        stable_matching <<< 1, n >>>(n, d_men, d_women, d_menacc, d_womenacc, d_menpre, d_matched, d_womenlock);
        hipDeviceSynchronize();
    }
}


int main()
{
    int n,i,j,k;
    int *d_matched;
    int *men, *women;
    int *menacc, *womenacc, *menpre, *womenlock;
    int *d_men, *d_women;
    int *d_menacc, *d_womenacc, *d_menpre, *d_womenlock;
    clock_t beg, end;
    double time_taken;

    scanf("%d",&n);
    men = (int *) malloc((n+1)*(n+1)*sizeof(int));
    women = (int *) malloc((n+1)*(n+1)*sizeof(int));
    menacc = (int *) malloc((n+1)*sizeof(int));
    womenacc = (int *) malloc((n+1)*sizeof(int));
    womenlock = (int *) malloc((n+1)*sizeof(int));
    menpre = (int *) malloc((n+1)*sizeof(int));

    hipMalloc(&d_men, (n+1)*(n+1)*sizeof(int));
    hipMalloc(&d_women, (n+1)*(n+1)*sizeof(int));
    hipMalloc(&d_menacc, (n+1)*sizeof(int));
    hipMalloc(&d_womenacc, (n+1)*sizeof(int));
    hipMalloc(&d_womenlock, (n+1)*sizeof(int));
    hipMalloc(&d_menpre, (n+1)*sizeof(int));
    hipMalloc(&d_matched, sizeof(int));

    initialize(menacc, womenacc, menpre, womenlock, n);

    beg = clock();
    for(i=1; i<=n; i++) {
        for(j=0; j<=n; j++) {
            scanf("%d", &men[i*(n+1) + j]);
        }
    }

    for(i=1; i<=n; i++) {
        for(j=0; j<=n; j++) {
            scanf("%d", &k);
            women[i*(n+1) + k] = j;
        }
    }
    end = clock();
    time_taken = ((double)(end-beg) * 1000000)/CLOCKS_PER_SEC;
    printf("read time : %f us, ", time_taken);

    hipMemcpy(d_men, men, (n+1)*(n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_women, women, (n+1)*(n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_menacc, menacc, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_womenlock, womenlock, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_womenacc, womenacc, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_menpre, menpre, (n+1)*sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    // uncomment this part to use kernel-2

    // int matched = 0;
    // while(!matched) {
    //     matched = 1;
    //     hipMemcpy(d_matched, &matched, sizeof(int), hipMemcpyHostToDevice);
    //     stable_matching <<< 1, n >>>(n, d_men, d_women, d_menacc, d_womenacc, d_menpre, d_matched, d_womenlock);
    //     hipMemcpy(&matched, d_matched, sizeof(int), hipMemcpyDeviceToHost);
    // }

    // kernel-3 implementation

    driver_function <<< 1, 1 >>>(n, d_men, d_women, d_menacc, d_womenacc, d_menpre, d_matched, d_womenlock);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(menacc, d_menacc, (n+1)*sizeof(int), hipMemcpyDeviceToHost);
    printf("compute time : %f us\n", milliseconds*1000);

    for(j=1;j<=n;j++)
        printf("%d %d\n", j, menacc[j]);

    free(men); free(women);
    free(menacc); free(womenacc); free(menpre); free(womenlock);
    hipFree(&d_men); hipFree(&d_women); hipFree(&d_matched);
    hipFree(&d_menacc); hipFree(&d_womenacc); hipFree(&d_menpre); hipFree(&d_womenlock);

    return 0;
}